#include "hip/hip_runtime.h"


__global__ void heightMapUpdateKernel(
    hipTextureObject_t in,
    hipTextureObject_t out,
    float *params,
    float *sensorToZUpFrameTf,
    float *zUpToSensorFrameTf)
{
    // Get the thread's unique x and y indices
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the indices are within the bounds of the image
    if (xIndex >= (int)params[DEPTH_INPUT_WIDTH] || yIndex >= (int)params[DEPTH_INPUT_HEIGHT])
        return;

    float currentAverageHeight = 0.0f;
    float averageHeightZ = 0.0f;
    float3 cellCenterInZUp = make_float3(0.0f, 0.0f, 0.5f);
    cellCenterInZUp.xy = indices_to_coordinate(make_int2(xIndex, yIndex),
                                               make_float2(0, 0), // params[HEIGHT_MAP_CENTER_X], params[HEIGHT_MAP_CENTER_Y]
                                               params[LOCAL_CELL_SIZE],
                                               params[LOCAL_CENTER_INDEX]);

    cellCenterInZUp.x += params[GRID_OFFSET_X];

    float halfCellWidth = params[LOCAL_CELL_SIZE] / 2.0f;
    float minX = cellCenterInZUp.x - halfCellWidth;
    float maxX = cellCenterInZUp.x + halfCellWidth;
    float minY = cellCenterInZUp.y - halfCellWidth;
    float maxY = cellCenterInZUp.y + halfCellWidth;

    int count = 0;
    int skip = (int)params[SEARCH_SKIP_SIZE];

    float3 cellCenterInSensor = transformPoint3D32_2(
        cellCenterInZUp,
        make_float3(sensorToZUpFrameTf[0], sensorToZUpFrameTf[1], sensorToZUpFrameTf[2]),
        make_float3(sensorToZUpFrameTf[4], sensorToZUpFrameTf[5], sensorToZUpFrameTf[6]),
        make_float3(sensorToZUpFrameTf[8], sensorToZUpFrameTf[9], sensorToZUpFrameTf[10]),
        make_float3(sensorToZUpFrameTf[3], sensorToZUpFrameTf[7], sensorToZUpFrameTf[11]));

    int2 projectedPoint;
    if (params[MODE] == 0) // Spherical Projection
    {
        projectedPoint = spherical_projection(cellCenterInSensor, params);
    }
    else if (params[MODE] == 1) // Perspective Projection
    {
        // Convert cellCenterInSensor to z-forward, x-right, y-down
        float3 cellCenterInSensorZfwd = make_float3(-cellCenterInSensor.y, -cellCenterInSensor.z, cellCenterInSensor.x);

        if (cellCenterInSensorZfwd.z < 0)
            return;

        projectedPoint = perspective_projection(cellCenterInSensorZfwd, params);
    }

    for (int pitch_count_offset = -((int)params[SEARCH_WINDOW_HEIGHT] / 2);
         pitch_count_offset < ((int)params[SEARCH_WINDOW_HEIGHT] / 2 + 1);
         pitch_count_offset += skip)
    {
        int pitch_count = projectedPoint.y + pitch_count_offset;
        for (int yaw_count_offset = -((int)params[SEARCH_WINDOW_WIDTH] / 2);
             yaw_count_offset < ((int)params[SEARCH_WINDOW_WIDTH] / 2 + 1);
             yaw_count_offset += skip)
        {
            int yaw_count = projectedPoint.x + yaw_count_offset;
            if ((yaw_count >= 0) && (yaw_count < (int)params[DEPTH_INPUT_WIDTH]) &&
                (pitch_count >= 0) && (pitch_count < (int)params[DEPTH_INPUT_HEIGHT]))
            {
                float depth = tex2D<float>(in, yaw_count, pitch_count) / 1000.0f;
                float3 queryPointInSensor;
                if (params[MODE] == 0) // Spherical
                {
                    queryPointInSensor = back_project_spherical(yaw_count, pitch_count, depth, params);
                }
                else if (params[MODE] == 1) // Perspective
                {
                    queryPointInSensor = back_project_perspective(make_int2(yaw_count, pitch_count), depth, params);
                }

                float3 queryPointInZUp = transformPoint3D32_2(
                    queryPointInSensor,
                    make_float3(sensorToZUpFrameTf[0], sensorToZUpFrameTf[1], sensorToZUpFrameTf[2]),
                    make_float3(sensorToZUpFrameTf[4], sensorToZUpFrameTf[5], sensorToZUpFrameTf[6]),
                    make_float3(sensorToZUpFrameTf[8], sensorToZUpFrameTf[9], sensorToZUpFrameTf[10]),
                    make_float3(sensorToZUpFrameTf[3], sensorToZUpFrameTf[7], sensorToZUpFrameTf[11]));

                if (queryPointInZUp.x > minX && queryPointInZUp.x < maxX &&
                    queryPointInZUp.y > minY && queryPointInZUp.y < maxY)
                {
                    // Remove outliers before averaging for a single cell
                    if (count > 1)
                    {
                        currentAverageHeight = averageHeightZ / (float)(count);
                        if (fabs(queryPointInZUp.z - currentAverageHeight) > 0.1f)
                        {
                            continue;
                        }
                    }
                    count++;
                    averageHeightZ += queryPointInZUp.z;
                }
            }
        }
    }

    if (count > 0)
    {
        averageHeightZ = averageHeightZ / (float)(count);
    }
    else
    {
        // This is slightly below the floor height of what we'll accept
        averageHeightZ = -params[HEIGHT_OFFSET];
    }
    averageHeightZ = clamp(averageHeightZ, params[MIN_CLAMP_HEIGHT], params[MAX_CLAMP_HEIGHT]);
    averageHeightZ += params[HEIGHT_OFFSET];

    // Write the result to the output texture
    uint4 outputValue = make_uint4(static_cast<int>((averageHeightZ) * params[HEIGHT_SCALING_FACTOR]), 0, 0, 0);
    surf2Dwrite(outputValue, out, yIndex * sizeof(uint4), xIndex);
}